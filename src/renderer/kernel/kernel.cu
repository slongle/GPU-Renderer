#include "hip/hip_runtime.h"
#ifndef _RENDER_KERNEL_CU_
#define _RENDER_KERNEL_CU_

#include "utility/hip/hip_runtime_api.h"
#include "utility/hip/hip_vector_types.h"
#include "utility/helper_functions.h"

// CUDA Runtime, includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <driver_functions.h>

#include "renderer/core/transform.h"
#include "renderer/core/renderer.h"
#include "renderer/core/sampling.h"
#include "renderer/core/camera.h"
#include "renderer/core/geometry.h"

#include "renderer/kernel/cudascene.h"
#include "renderer/kernel/cudarenderer.h"

struct PathState {
    // Film position
    uint x, y;
    // Pipline information
    uint nSample;
    uint state;
    // Path information
    Spectrum L;
    uint seed;
    Spectrum throughput;
    Ray ray;
    bool specular;
    int bounce;
    bool hit;
    Interaction inter;
};

struct Queue {
    __device__ __host__
    bool isEmpty() {
        return queue[l] == -1;
    }

    __device__ __host__
    void push(int v) {
        queue[r] = v;
        r++;
        if (r == size) {
            r = 0;
        }
    }

    __device__ __host__
    int pop() {
        int ret = queue[l];
        queue[l] = -1;
        l++;
        if (l == size) {
            l = 0;
        }
        return ret;
    }

    int l, r;
    int size;
    int* queue;
};

CUDAScene* hst_scene;
CUDAScene* dev_scene;
Camera* hst_camera;
Camera* dev_camera;
Integrator* hst_integrator;
Integrator* dev_integrator;
CUDARenderer* hst_renderer;
CUDARenderer* dev_renderer;

PathState* pathStates;
Queue* newPathRequest;
Queue* intersectRequest;
Queue* materialRequest;
Queue* neeRequest;

unsigned int frame = 0;

__global__
void InitState(
    PathState* pathStates, 
    int pathSize, int width, 
    Queue* newPathRequest,    
    Queue* intersectRequest,
    Queue* materialRequest, 
    Queue* neeRequest) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < pathSize) {
        int y = index / width;
        int x = index % width;
        pathStates[index].x = x;
        pathStates[index].y = y;
        pathStates[index].state = 0;
        newPathRequest->queue[index] = intersectRequest->queue[index] 
            = materialRequest->queue[index] = neeRequest->queue[index] = -1;
    }
}

extern "C"
void cudaInit(std::shared_ptr<Renderer> renderer) {
    // Move Scene Data
    Scene* scene = &(renderer->m_scene);
    scene->Preprocess();
    hst_scene = new CUDAScene(scene);

    // Move TriangleMesh Data
    int triangleMeshNum = hst_scene->m_triangleMeshNum;
    for (int i = 0; i < triangleMeshNum; i++) {
        int triangleNum = scene->m_triangleMeshes[i].m_triangleNum;
        hipMalloc(&hst_scene->m_triangleMeshes[i].m_indices, 3 * triangleNum * sizeof(int));
        hipMemcpy(hst_scene->m_triangleMeshes[i].m_indices, scene->m_triangleMeshes[i].m_indices,
            3 * triangleNum * sizeof(int), hipMemcpyHostToDevice);
        int vertexNum = scene->m_triangleMeshes[i].m_vertexNum;
        hipMalloc(&hst_scene->m_triangleMeshes[i].m_P, vertexNum * sizeof(Point3f));
        hipMemcpy(hst_scene->m_triangleMeshes[i].m_P, scene->m_triangleMeshes[i].m_P,
            vertexNum * sizeof(Point3f), hipMemcpyHostToDevice);

        if (scene->m_triangleMeshes[i].m_N) {
            hipMalloc(&hst_scene->m_triangleMeshes[i].m_N, vertexNum * sizeof(Normal3f));
            hipMemcpy(hst_scene->m_triangleMeshes[i].m_N, scene->m_triangleMeshes[i].m_N,
                vertexNum * sizeof(Normal3f), hipMemcpyHostToDevice);
        }

        if (scene->m_triangleMeshes[i].m_UV) {
            hipMalloc(&hst_scene->m_triangleMeshes[i].m_UV, vertexNum * sizeof(Point2f));
            hipMemcpy(hst_scene->m_triangleMeshes[i].m_UV, scene->m_triangleMeshes[i].m_UV,
                vertexNum * sizeof(Point2f), hipMemcpyHostToDevice);
        }
    }
    TriangleMesh* triangleMeshGPUPtr;
    hipMalloc(&triangleMeshGPUPtr, triangleMeshNum * sizeof(TriangleMesh));
    hipMemcpy(triangleMeshGPUPtr, hst_scene->m_triangleMeshes, triangleMeshNum * sizeof(TriangleMesh), hipMemcpyHostToDevice);
    hst_scene->m_triangleMeshes = triangleMeshGPUPtr;

    // Move Triangle Data
    int triangleNum = scene->m_triangles.size();
    for (int i = 0; i < triangleNum; i++) {
        int meshID = scene->m_triangles[i].m_triangleMeshID;
        scene->m_triangles[i].m_triangleMeshPtr = triangleMeshGPUPtr + meshID;
    }
    hipMalloc(&hst_scene->m_triangles, sizeof(Triangle) * triangleNum);
    hipMemcpy(hst_scene->m_triangles, scene->m_triangles.data(),
        sizeof(Triangle) * triangleNum, hipMemcpyHostToDevice);


    // Move Material Data
    int materialNum = scene->m_materials.size();
    hipMalloc(&hst_scene->m_materials, sizeof(Material) * materialNum);
    hipMemcpy(hst_scene->m_materials, scene->m_materials.data(),
        sizeof(Material) * materialNum, hipMemcpyHostToDevice);

    // Move Light Data
    int lightNum = scene->m_lights.size();
    hipMalloc(&hst_scene->m_lights, sizeof(Light) * lightNum);
    hipMemcpy(hst_scene->m_lights, scene->m_lights.data(),
        sizeof(Light) * lightNum, hipMemcpyHostToDevice);

    // Move Primitive Data
    int primitiveNum = scene->m_primitives.size();
    hipMalloc(&hst_scene->m_primitives, sizeof(Primitive) * primitiveNum);
    hipMemcpy(hst_scene->m_primitives, scene->m_primitives.data(),
        sizeof(Primitive) * primitiveNum, hipMemcpyHostToDevice);

    // Move BVH Data   
    CUDABVH* hst_bvh = new CUDABVH();    
    hst_bvh->m_maxPrimsInNode = scene->m_shapeBvh->m_maxPrimsInNode;
    hst_bvh->m_totalNodes = scene->m_shapeBvh->m_totalNodes;
    hst_bvh->m_splitMethod = static_cast<CUDABVH::SplitMethod>(scene->m_shapeBvh->m_splitMethod);
    hipMalloc(&hst_bvh->m_primitives, sizeof(Primitive) * scene->m_shapeBvh->m_primitives.size());
    hipMemcpy(hst_bvh->m_primitives, scene->m_shapeBvh->m_primitives.data(),
        sizeof(Primitive) * scene->m_shapeBvh->m_primitives.size(), hipMemcpyHostToDevice);    
    hipMalloc(&hst_bvh->m_nodes, sizeof(LinearBVHNode) * scene->m_shapeBvh->m_totalNodes);
    hipMemcpy(hst_bvh->m_nodes, scene->m_shapeBvh->m_nodes,
        sizeof(LinearBVHNode) * scene->m_shapeBvh->m_totalNodes, hipMemcpyHostToDevice);
    memcpy(&hst_scene->m_bvh, hst_bvh, sizeof(CUDABVH));

    // Move cudaScene Data
    hipMalloc(&dev_scene, sizeof(CUDAScene));
    hipMemcpy(dev_scene, hst_scene, sizeof(CUDAScene), hipMemcpyHostToDevice);    

    // Move Camera Data
    hst_camera = &(renderer->m_camera);
    Film& film = hst_camera->m_film;
    hipMalloc(&film.m_bitmap, film.m_resolution.x * film.m_resolution.y * 3 * sizeof(Float));
    hipMemset(film.m_bitmap, 0, film.m_resolution.x * film.m_resolution.y * 3 * sizeof(Float));
    hipMalloc(&film.m_sampleNum, film.m_resolution.x * film.m_resolution.y * sizeof(unsigned int));
    hipMemset(film.m_sampleNum, 0, film.m_resolution.x * film.m_resolution.y * sizeof(unsigned int));
    hipMalloc(&dev_camera, sizeof(Camera));
    checkCudaErrors(hipMemcpy(dev_camera, hst_camera, sizeof(Camera), hipMemcpyHostToDevice));

    // Move Integrator
    hst_integrator = &(renderer->m_integrator);
    hipMalloc(&dev_integrator, sizeof(Integrator));
    hipMemcpy(dev_integrator, hst_integrator, sizeof(Integrator), hipMemcpyHostToDevice);

    hst_renderer = new CUDARenderer(dev_integrator, dev_camera, dev_scene);
    hipMalloc(&dev_renderer, sizeof(CUDARenderer));
    checkCudaErrors(hipMemcpy(dev_renderer, hst_renderer, sizeof(CUDARenderer), hipMemcpyHostToDevice));
    
    

    int pathSize = film.m_resolution.x* film.m_resolution.y;    
    hipMallocManaged(&pathStates, sizeof(PathState) * pathSize);

    hipMallocManaged(&newPathRequest, sizeof(Queue));
    hipMallocManaged(&newPathRequest->queue, sizeof(int) * pathSize);
    newPathRequest->size = pathSize;
    newPathRequest->l = newPathRequest->r = 0;

    hipMallocManaged(&intersectRequest, sizeof(Queue));
    hipMallocManaged(&intersectRequest->queue, sizeof(int) * pathSize);
    intersectRequest->size = pathSize;
    intersectRequest->l = intersectRequest->r = 0;

    hipMallocManaged(&materialRequest, sizeof(Queue));
    hipMallocManaged(&materialRequest->queue, sizeof(int) * pathSize);
    materialRequest->size = pathSize;
    materialRequest->l = materialRequest->r = 0;

    hipMallocManaged(&neeRequest, sizeof(Queue));
    hipMallocManaged(&neeRequest->queue, sizeof(int) * pathSize);
    neeRequest->size = pathSize;
    neeRequest->l = neeRequest->r = 0;

    int blockSize = 16;
    int gridSize = (pathSize + blockSize - 1) / blockSize;    
    InitState << <gridSize, blockSize >> > (
        pathStates, pathSize, film.m_resolution.x,
        newPathRequest, intersectRequest, materialRequest, neeRequest);
}

__global__
void logic(CUDARenderer* renderer, int pathSize,
    PathState* pathStates, Queue* intersectRequest, Queue* newPathRequest, Queue* materialRequest, Queue* neeRequest) {    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < pathSize) {
        CUDAScene* scene = renderer->m_scene;
        Integrator* integrator = renderer->m_integrator;
        Camera* camera = renderer->m_camera;
        Film* film = &camera->m_film;
        PathState& path = pathStates[index];
        if (path.state == 0) {
            newPathRequest->push(index);
        }
        else if (path.state == 1) {
            intersectRequest->push(index);
        }
        else if (path.state == 2) {
            const Primitive& primitive = scene->m_primitives[path.inter.m_primitiveID];
            const Material& material = scene->m_materials[primitive.m_materialID];
            if (path.bounce == 0 || path.specular) {
                if (primitive.m_lightID != -1) {
                    int lightID = primitive.m_lightID;
                    const Light& light = scene->m_lights[lightID];
                    if (Dot(path.inter.m_shadingN, path.inter.m_wo) > 0) {
                        path.L += path.throughput * light.m_L;
                    }
                }
            }
            path.state = 3;
        }
        else if (path.state == 3) {
            neeRequest->push(index);
        }
        else if (path.state == 4) {
            materialRequest->push(index);
        }
        else if (path.state == 5) {
            if (path.throughput.Max() < 1 && path.bounce > 3) {
                Float q = max((Float).05, 1 - path.throughput.Max());
                if (NextRandom(path.seed) < q) {
                    path.state = 0;
                    camera->m_film.AddSample(path.x, path.y, path.L);
                }                
                path.throughput /= 1 - q;
            }    
            if (path.state != 0) {
                if (path.bounce < integrator->m_maxDepth) {
                    path.state = 0;
                    camera->m_film.AddSample(path.x, path.y, path.L);
                }
                else {
                    path.state = 1;
                    path.ray = Ray(path.inter.m_p, path.inter.m_wi);
                }
            }
        }

    }
}

__global__
void newPath(int pathSize) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < pathSize) {

    }
}

__global__
void materialEvaluate(int pathSize) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < pathSize) {

    }
}

__global__
void nee(int pathSize) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < pathSize) {

    }
}


void render(CUDARenderer* renderer) {
    Film& film = hst_camera->m_film;
    int pathSize = film.m_resolution.x * film.m_resolution.y;
    int blockSize = 16;
    int gridSize = (pathSize + blockSize - 1) / blockSize;

    logic << <gridSize, blockSize >> > (renderer,
        pathSize, pathStates, intersectRequest, newPathRequest, materialRequest, neeRequest);
    newPath << <gridSize, blockSize >> > (pathSize);
    materialEvaluate << <gridSize, blockSize >> > (pathSize);
    nee<< <gridSize, blockSize >> > (pathSize);
}

typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

inline __device__
Float PowerHeuristic(int nf, Float fPdf, int ng, Float gPdf) {
    Float f = nf * fPdf, g = ng * gPdf;
    return (f * f) / (f * f + g * g);
}

inline __device__
Spectrum NextEventEstimate(const CUDAScene& scene, const Interaction& inter, unsigned int& seed, Point3f& pLight) 
{    
    const Primitive& primitive = scene.m_primitives[inter.m_primitiveID];
    const Material& material = scene.m_materials[primitive.m_materialID];

    Spectrum est;
    
    // Sample one of lights
    int lightID = min(scene.m_lightNum - 1, int(NextRandom(seed) * scene.m_lightNum));
    Float lightChoosePdf = Float(1) / scene.m_lightNum;
    const Light& light = scene.m_lights[lightID];

    // Light Sampling
    {
        // Light Sample Li
        const Triangle& triangle = scene.m_triangles[light.m_shapeID];
        Float lightSamplePdf;
        Interaction lightSample = triangle.Sample(&lightSamplePdf, seed);
        pLight = lightSample.m_p;
        lightSamplePdf *= (lightSample.m_p - inter.m_p).SqrLength() / 
            AbsDot(-Normalize(lightSample.m_p - inter.m_p), lightSample.m_shadingN);

        // Visibility test
        Ray testRay = inter.SpawnRayTo(lightSample);
        bool hit = scene.Intersect(testRay);

        if (!hit) {
            Vector3f d = Normalize(lightSample.m_p - inter.m_p);
            // Get Le
            Spectrum Le(0.);
            if (Dot(-d, lightSample.m_shadingN) > 0) {
                Le = light.m_L;
            }

            // BSDF Sample
            Normal3f n = inter.m_shadingN;
            Float bsdfPdf;
            Spectrum cosBSDF;
            cosBSDF = material.F(n, inter.m_wo, d, &bsdfPdf);

            // Contribution
            if (light.isDelta()) {
                est += Le * cosBSDF / lightSamplePdf;
            }
            else {
                Float weight = PowerHeuristic(1, lightSamplePdf, 1, bsdfPdf);                
                est += Le * cosBSDF * weight / lightSamplePdf;
            }
        }
    }

    // BSDF Sampling
    if (!light.isDelta()) {

        // BSDF Sample
        Normal3f n = inter.m_shadingN;
        Float bsdfPdf;
        Spectrum cosBSDF;
        Vector3f wi;
        cosBSDF = material.Sample(n, inter.m_wo, &wi, &bsdfPdf, seed);

        // Light Sample
        const Triangle& triangle = scene.m_triangles[light.m_shapeID];

        Point3f origin = inter.m_p + wi * Epsilon;
        Ray testRay(origin, wi);
        Interaction lightInter;
        bool hit = scene.IntersectP(testRay, &lightInter);

        if (hit && scene.m_primitives[lightInter.m_primitiveID].m_lightID != -1)
        {
            Float lightSamplePdf;
            lightSamplePdf = (lightInter.m_p - inter.m_p).SqrLength() /
                (AbsDot(-wi, lightInter.m_shadingN) * triangle.Area());
            pLight = lightInter.m_p;

            // Get Le            
            Spectrum Le(0.);
            if (Dot(-wi, lightInter.m_shadingN) > 0) {
                Le = light.m_L;
            }

            Float weight = PowerHeuristic(1, bsdfPdf, 1, lightSamplePdf);
            est += Le * cosBSDF * weight / bsdfPdf;            
        }
    }

    return est / lightChoosePdf;
}

inline __device__
Spectrum SampleMaterial(const CUDAScene& scene, Interaction& inter, unsigned int& seed) {
    const Primitive& primitive = scene.m_primitives[inter.m_primitiveID];
    const Material& material = scene.m_materials[primitive.m_materialID];
    
    Normal3f n = inter.m_shadingN;
    
    Float bsdfPdf;
    Spectrum cosBSDF = material.Sample(n, inter.m_wo, &inter.m_wi, &bsdfPdf, seed);

    return cosBSDF / bsdfPdf;
}

__global__ void
d_render(uint* d_output, uint imageW, uint imageH, unsigned int frame, CUDARenderer* renderer)
{
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    Integrator* integrator = renderer->m_integrator;
    Camera* camera = renderer->m_camera;
    CUDAScene* scene = renderer->m_scene;
    //printf("%d\n", scene->m_bvh.m_primitives);

    uint index = y * imageW + x;
    if ((x >= imageW) || (y >= imageH)) return;

    Spectrum L(0);    
    uint seed = InitRandom(index, frame);
    Spectrum throughput(1);
    Ray ray = camera->GenerateRay(Point2f(x + NextRandom(seed), y + NextRandom(seed)));
    bool specular = false;
    int bounce;
    for (bounce = 0; bounce < integrator->m_maxDepth; bounce++) {

        // find intersection with scene
        Interaction inter;
        bool hit = scene->IntersectP(ray, &inter);
        if (!hit) {
            break;
        }

        const Primitive& primitive = scene->m_primitives[inter.m_primitiveID];
        const Material& material = scene->m_materials[primitive.m_materialID];
        if (bounce == 0 || specular) {
            if (primitive.m_lightID != -1) {
                int lightID = primitive.m_lightID;
                const Light& light = scene->m_lights[lightID];
                if (Dot(inter.m_shadingN, inter.m_wo) > 0) {
                    L += throughput * light.m_L;
                }
            }
        }

        // render normal
        //L = Spectrum(inter.m_geometryN);
        //break;

        if (throughput.isBlack()) {
            break;
        }

        // direct light
        Point3f pLight;
        if (!material.isDelta()) {
            L += throughput * NextEventEstimate(*scene, inter, seed, pLight);
            specular = false;
        }
        else {
            specular = true;
        }

        // calculate BSDF
        throughput *= SampleMaterial(*scene, inter, seed);
        //break;

        // indirect light                    
        if (throughput.Max() < 1 && bounce > 3) {
            Float q = max((Float).05, 1 - throughput.Max());
            if (NextRandom(seed) < q) break;
            throughput /= 1 - q;
        }

        ray = Ray(inter.m_p, inter.m_wi);
    }
    camera->m_film.AddSample(x, y, L);
    L = camera->m_film.GetPixelSpectrum(index);


    // write output color
    if (d_output) {
        SpectrumToUnsignedChar(L, (unsigned char*)&d_output[(imageH - y - 1) * imageW + x], 4);
    }
    else {
        //printf("0");
    }

}

extern "C"
void freeCudaBuffers()
{
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint * d_output, uint imageW, uint imageH)
{    
    d_render << <gridSize, blockSize >> > (d_output, imageW, imageH, frame, dev_renderer);
    //checkCudaErrors(hipDeviceSynchronize());
    frame++;
}

extern "C"
void copyInvViewMatrix(float* invViewMatrix, size_t sizeofMatrix)
{
    //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}

extern "C"
void gpu_render(std::shared_ptr<Renderer> renderer) {
    auto iDivUp = [](unsigned int a, unsigned int b)->unsigned int {
        return (a + b - 1) / b;
    };

    cudaInit(renderer);

    int width = renderer->m_camera.m_film.m_resolution.x;
    int height = renderer->m_camera.m_film.m_resolution.y;
    Camera* camera = &renderer->m_camera;
    Film film = camera->m_film;
    
    dim3 blockSize{ 16, 16 };
    dim3 gridSize{ iDivUp(width, blockSize.x), iDivUp(height, blockSize.y) };
    for (unsigned int i = 0; i < renderer->m_integrator.m_nSample; i++) {
        d_render << <gridSize, blockSize >> > (NULL, width, height, i, dev_renderer);
    }
    checkCudaErrors(hipDeviceSynchronize());

    film.m_bitmap = new Float[film.m_resolution.x * film.m_resolution.y * 3];
    film.m_sampleNum = new unsigned int[film.m_resolution.x * film.m_resolution.y];
    memset(film.m_bitmap, 0, sizeof(Float) * film.m_resolution.x * film.m_resolution.y * 3);
    memset(film.m_sampleNum, 0, sizeof(unsigned int) * film.m_resolution.x * film.m_resolution.y);    
    checkCudaErrors(hipMemcpy(film.m_bitmap, hst_camera->m_film.m_bitmap, sizeof(Float) * film.m_resolution.x * film.m_resolution.y * 3, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(film.m_sampleNum, hst_camera->m_film.m_sampleNum, sizeof(unsigned int) * film.m_resolution.x * film.m_resolution.y, hipMemcpyDeviceToHost));
    //hipDeviceSynchronize();
    film.Output("GPU-");     
}

#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
