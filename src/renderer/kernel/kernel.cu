#include "hip/hip_runtime.h"
#ifndef _RENDER_KERNEL_CU_
#define _RENDER_KERNEL_CU_

#include "utility/hip/hip_runtime_api.h"
#include "utility/hip/hip_vector_types.h"
#include "utility/helper_functions.h"

// CUDA Runtime, includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <driver_functions.h>

#include "renderer/core/transform.h"
#include "renderer/core/renderer.h"
#include "renderer/core/sampling.h"
#include "renderer/core/camera.h"
#include "renderer/core/geometry.h"

#include "renderer/kernel/cudascene.h"
#include "renderer/kernel/cudarenderer.h"

CUDAScene* hst_scene;
CUDAScene* dev_scene;
Camera* hst_camera;
Camera* dev_camera;
Integrator* hst_integrator;
Integrator* dev_integrator;
CUDARenderer* hst_renderer;
CUDARenderer* dev_renderer;

//int frame = 0;

extern "C"
void cudaInit(std::shared_ptr<Renderer> renderer) {
    // Move Scene Data
    Scene* scene = &(renderer->m_scene);
    hst_scene = new CUDAScene(scene);

    // Move TriangleMesh Data
    int triangleMeshNum = hst_scene->m_triangleMeshNum;
    for (int i = 0; i < triangleMeshNum; i++) {
        int triangleNum = scene->m_triangleMeshes[i].m_triangleNum;
        hipMalloc(&hst_scene->m_triangleMeshes[i].m_indices, 3 * triangleNum * sizeof(int));
        hipMemcpy(hst_scene->m_triangleMeshes[i].m_indices, scene->m_triangleMeshes[i].m_indices,
            3 * triangleNum * sizeof(int), hipMemcpyHostToDevice);
        int vertexNum = scene->m_triangleMeshes[i].m_vertexNum;
        hipMalloc(&hst_scene->m_triangleMeshes[i].m_P, vertexNum * sizeof(Point3f));
        hipMemcpy(hst_scene->m_triangleMeshes[i].m_P, scene->m_triangleMeshes[i].m_P,
            vertexNum * sizeof(Point3f), hipMemcpyHostToDevice);

        if (scene->m_triangleMeshes[i].m_N) {
            hipMalloc(&hst_scene->m_triangleMeshes[i].m_N, vertexNum * sizeof(Normal3f));
            hipMemcpy(hst_scene->m_triangleMeshes[i].m_N, scene->m_triangleMeshes[i].m_N,
                vertexNum * sizeof(Normal3f), hipMemcpyHostToDevice);
        }

        if (scene->m_triangleMeshes[i].m_UV) {
            hipMalloc(&hst_scene->m_triangleMeshes[i].m_UV, vertexNum * sizeof(Point2f));
            hipMemcpy(hst_scene->m_triangleMeshes[i].m_UV, scene->m_triangleMeshes[i].m_UV,
                vertexNum * sizeof(Point2f), hipMemcpyHostToDevice);
        }
    }
    TriangleMesh* triangleMeshGPUPtr;
    hipMalloc(&triangleMeshGPUPtr, triangleMeshNum * sizeof(TriangleMesh));
    hipMemcpy(triangleMeshGPUPtr, hst_scene->m_triangleMeshes, triangleMeshNum * sizeof(TriangleMesh), hipMemcpyHostToDevice);
    hst_scene->m_triangleMeshes = triangleMeshGPUPtr;

    // Move Triangle Data
    int triangleNum = scene->m_triangles.size();
    for (int i = 0; i < triangleNum; i++) {
        int meshID = scene->m_triangles[i].m_triangleMeshID;
        scene->m_triangles[i].m_triangleMeshPtr = triangleMeshGPUPtr + meshID;
    }
    hipMalloc(&hst_scene->m_triangles, sizeof(Triangle) * triangleNum);
    hipMemcpy(hst_scene->m_triangles, scene->m_triangles.data(),
        sizeof(Triangle) * triangleNum, hipMemcpyHostToDevice);


    // Move Material Data
    int materialNum = scene->m_materials.size();
    hipMalloc(&hst_scene->m_materials, sizeof(Material) * materialNum);
    hipMemcpy(hst_scene->m_materials, scene->m_materials.data(),
        sizeof(Material) * materialNum, hipMemcpyHostToDevice);

    // Move Light Data
    int lightNum = scene->m_lights.size();
    hipMalloc(&hst_scene->m_lights, sizeof(Light) * lightNum);
    hipMemcpy(hst_scene->m_lights, scene->m_lights.data(),
        sizeof(Light) * lightNum, hipMemcpyHostToDevice);

    // Move Primitive Data
    int primitiveNum = scene->m_primitives.size();
    hipMalloc(&hst_scene->m_primitives, sizeof(Primitive) * primitiveNum);
    hipMemcpy(hst_scene->m_primitives, scene->m_primitives.data(),
        sizeof(Primitive) * primitiveNum, hipMemcpyHostToDevice);

    // Move cudaScene Data
    hipMalloc(&dev_scene, sizeof(CUDAScene));
    hipMemcpy(dev_scene, hst_scene, sizeof(CUDAScene), hipMemcpyHostToDevice);

    // Move Camera Data
    hst_camera = &(renderer->m_camera);
    Film& film = hst_camera->m_film;
    hipMalloc(&film.m_bitmap, film.m_resolution.x * film.m_resolution.y * sizeof(unsigned char));
    hipMemset(film.m_bitmap, 0, film.m_resolution.x * film.m_resolution.y * sizeof(unsigned char));
    hipMalloc(&dev_camera, sizeof(Camera));
    hipMemcpy(dev_camera, hst_camera, sizeof(Camera), hipMemcpyHostToDevice);

    // Move Integrator
    hst_integrator = &(renderer->m_integrator);
    hipMalloc(&dev_integrator, sizeof(Integrator));
    hipMemcpy(dev_integrator, hst_integrator, sizeof(Integrator), hipMemcpyHostToDevice);

    hst_renderer = new CUDARenderer(dev_integrator, dev_camera, dev_scene);
    hipMalloc(&dev_renderer, sizeof(CUDARenderer));
    hipMemcpy(dev_renderer, hst_renderer, sizeof(CUDARenderer), hipMemcpyHostToDevice);
}

typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

/*struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float* tnear, float* tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}*/

__global__ void
d_render(uint* d_output, uint imageW, uint imageH, int frame, CUDARenderer* renderer)
{
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    Integrator* integrator = renderer->m_integrator;
    Camera* camera = renderer->m_camera;
    CUDAScene* scene = renderer->m_scene;

    uint index = y * imageW + x;
    //if (x != 123 || y != 234) return;
    /*
    if (index == 0) {
        printf("%d\n", scene->m_triangleMeshNum);
        printf("%d\n", scene->m_triangleMeshes[0].m_triangleNum);
        for (int i = 0; i < scene->m_triangleMeshes[0].m_triangleNum * 3; i++) {
            printf("%d ", scene->m_triangleMeshes[0].m_indices[i]);
        }
        printf("\n");
        //for (int i = 0; i < a->f.size(); i++) {
            //printf("%d\n", a->f[i]);
        //}
        Vector3f v(1, 2, 3);
        printf("%f %f %f\n", v.x, v.y, v.z);
        Float len = v.Length();
        printf("%f\n", len);
    }*/
    Float znear = 1e-2;

    if ((x >= imageW) || (y >= imageH)) return;

    uint seed = InitRandom(index, frame);
    Spectrum L(0);
    Spectrum throughput(1);

    Ray ray = camera->GenerateRay(Point2f(x + NextRandom(seed), y + NextRandom(seed)));
    /*
    Interaction interaction;
    bool hit = scene->IntersectP(ray, &interaction);
    L = Spectrum(interaction.m_geometryN);        
    */

    

    /*Point3f pHit1 = interaction.m_p;
    Point3f pCamera1 = camera->m_worldToCamera(pHit1);
    Point3f pCameraFilm1(pCamera1.x / pCamera1.z * znear, pCamera1.y / pCamera1.z * znear, znear);
    Point3f pFilm1 = camera->m_cameraToRaster(pCameraFilm1);
    
    printf("%f %f %f\n", pFilm1.x, pFilm1.y, pFilm1.z);

    Material& material = scene->m_materials[interaction.m_primitiveID];
    Vector3f wi = CosineSampleHemisphere(seed);
    Normal3f n = interaction.m_geometryN;
    Vector3f w = fabs(n.x) < 0.5 ? Vector3f(1, 0, 0) : fabs(n.y) < 0.5 ? Vector3f(0, 1, 0) : Vector3f(0, 0, 1);
    Vector3f u = Normalize(Cross(Vector3f(n), w));
    Vector3f v = Cross(Vector3f(n), u);
    interaction.m_wi = Normalize(Vector3f(n) * wi.z + u * wi.x + v * wi.y);

    Point3f pHit2 = pHit1 + interaction.m_wi * 0.1;
    Point3f pCamera2 = camera->m_worldToCamera(pHit2);
    Point3f pCameraFilm2(pCamera2.x / pCamera2.z * znear, pCamera2.y / pCamera2.z * znear, znear);
    Point3f pFilm2 = camera->m_cameraToRaster(pCameraFilm2);

    printf("%f %f %f\n", pFilm2.x, pFilm2.y, pFilm2.z);
    */
    
    //camera->m_worldToCamera()

    for (int i = 0; i < integrator->m_maxDepth; i++) {

        // find intersection with scene
        Interaction interaction;
        bool hit = scene->IntersectP(ray, &interaction);

        if (!hit){
            break;
        }

        Primitive& primitive = scene->m_primitives[interaction.m_primitiveID];
        Material& material = scene->m_materials[primitive.m_materialID];
        // direct light
        if (primitive.m_lightID != -1) {
            int lightID = scene->m_primitives[interaction.m_primitiveID].m_lightID;
            if (throughput.r != 1 || throughput.g != 1 || throughput.b != 1) {
                int a = 0;
                a++;
            }
            L += throughput * scene->m_lights[lightID].m_L;
        }
        int lightID = min(scene->m_lightNum - 1, int(NextRandom(seed) * scene->m_lightNum));
        Light& light = scene->m_lights[lightID];
        Triangle& triangle = scene->m_triangles[light.m_shapeID];
        Float pdf;
        Interaction inter = triangle.Sample(&pdf, seed);
        Ray testRay(interaction.m_p + interaction.m_geometryN * Epsilon, inter.m_p - interaction.m_p, 1 - 2 * Epsilon);
        hit = scene->Intersect(testRay);
        if (!hit) {
            L += throughput * material.m_Kd * light.m_L / pdf;
        }


        // calculate BSDF
        Vector3f wi = CosineSampleHemisphere(seed);
        Normal3f n = interaction.m_geometryN;
        Vector3f w = fabs(n.x) < 0.5 ? Vector3f(1, 0, 0) : fabs(n.y) < 0.5 ? Vector3f(0, 1, 0) : Vector3f(0, 0, 1);
        Vector3f u = Normalize(Cross(Vector3f(n), w));
        Vector3f v = Cross(Vector3f(n), u);
        interaction.m_wi = Normalize(Vector3f(n)* wi.z + u * wi.x + v * wi.y);

        // indirect light
        if (i > 3) {
            Float q = min(Float(0.95), throughput.Max());
            if (NextRandom(seed) >= q) {
                break;
            }
            throughput /= q;
        }

        ray.o = interaction.m_p + interaction.m_geometryN * Epsilon;
        ray.d = interaction.m_wi;
        ray.tMax = Infinity;
    }
    

    // write output color
    SpectrumToUnsignedChar(L, (unsigned char*)&d_output[(imageH - y - 1) * imageW + x], 4);

}

extern "C"
void freeCudaBuffers()
{
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint * d_output, uint imageW, uint imageH)
{
    d_render << <gridSize, blockSize >> > (d_output, imageW, imageH, 0, dev_renderer);
    //hipDeviceSynchronize();
    //frame++;
    //exit(0);
}

extern "C"
void copyInvViewMatrix(float* invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}


#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
