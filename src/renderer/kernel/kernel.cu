#include "hip/hip_runtime.h"
#ifndef _RENDER_KERNEL_CU_
#define _RENDER_KERNEL_CU_

#include "utility/hip/hip_runtime_api.h"
#include "utility/hip/hip_vector_types.h"
#include "utility/helper_functions.h"

// CUDA Runtime, includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <driver_functions.h>

#include "renderer/core/transform.h"
#include "renderer/core/renderer.h"
#include "renderer/core/sampling.h"
#include "renderer/core/camera.h"
#include "renderer/core/geometry.h"

#include "renderer/kernel/cudascene.h"
#include "renderer/kernel/cudarenderer.h"

CUDAScene* hst_scene;
CUDAScene* dev_scene;
Camera* hst_camera;
Camera* dev_camera;
Integrator* hst_integrator;
Integrator* dev_integrator;
CUDARenderer* hst_renderer;
CUDARenderer* dev_renderer;

int frame = 0;

extern "C"
void cudaInit(std::shared_ptr<Renderer> renderer) {
    // Move Scene Data
    Scene* scene = &(renderer->m_scene);
    hst_scene = new CUDAScene(scene);

    // Move TriangleMesh Data
    int triangleMeshNum = hst_scene->m_triangleMeshNum;
    for (int i = 0; i < triangleMeshNum; i++) {
        int triangleNum = scene->m_triangleMeshes[i].m_triangleNum;
        hipMalloc(&hst_scene->m_triangleMeshes[i].m_indices, 3 * triangleNum * sizeof(int));
        hipMemcpy(hst_scene->m_triangleMeshes[i].m_indices, scene->m_triangleMeshes[i].m_indices,
            3 * triangleNum * sizeof(int), hipMemcpyHostToDevice);
        int vertexNum = scene->m_triangleMeshes[i].m_vertexNum;
        hipMalloc(&hst_scene->m_triangleMeshes[i].m_P, vertexNum * sizeof(Point3f));
        hipMemcpy(hst_scene->m_triangleMeshes[i].m_P, scene->m_triangleMeshes[i].m_P,
            vertexNum * sizeof(Point3f), hipMemcpyHostToDevice);

        if (scene->m_triangleMeshes[i].m_N) {
            hipMalloc(&hst_scene->m_triangleMeshes[i].m_N, vertexNum * sizeof(Normal3f));
            hipMemcpy(hst_scene->m_triangleMeshes[i].m_N, scene->m_triangleMeshes[i].m_N,
                vertexNum * sizeof(Normal3f), hipMemcpyHostToDevice);
        }

        if (scene->m_triangleMeshes[i].m_UV) {
            hipMalloc(&hst_scene->m_triangleMeshes[i].m_UV, vertexNum * sizeof(Point2f));
            hipMemcpy(hst_scene->m_triangleMeshes[i].m_UV, scene->m_triangleMeshes[i].m_UV,
                vertexNum * sizeof(Point2f), hipMemcpyHostToDevice);
        }
    }
    TriangleMesh* triangleMeshGPUPtr;
    hipMalloc(&triangleMeshGPUPtr, triangleMeshNum * sizeof(TriangleMesh));
    hipMemcpy(triangleMeshGPUPtr, hst_scene->m_triangleMeshes, triangleMeshNum * sizeof(TriangleMesh), hipMemcpyHostToDevice);
    hst_scene->m_triangleMeshes = triangleMeshGPUPtr;

    // Move Triangle Data
    int triangleNum = scene->m_triangles.size();
    for (int i = 0; i < triangleNum; i++) {
        int meshID = scene->m_triangles[i].m_triangleMeshID;
        scene->m_triangles[i].m_triangleMeshPtr = triangleMeshGPUPtr + meshID;
    }
    hipMalloc(&hst_scene->m_triangles, sizeof(Triangle) * triangleNum);
    hipMemcpy(hst_scene->m_triangles, scene->m_triangles.data(),
        sizeof(Triangle) * triangleNum, hipMemcpyHostToDevice);


    // Move Material Data
    int materialNum = scene->m_materials.size();
    hipMalloc(&hst_scene->m_materials, sizeof(Material) * materialNum);
    hipMemcpy(hst_scene->m_materials, scene->m_materials.data(),
        sizeof(Material) * materialNum, hipMemcpyHostToDevice);

    // Move Light Data
    int lightNum = scene->m_lights.size();
    hipMalloc(&hst_scene->m_lights, sizeof(Light) * lightNum);
    hipMemcpy(hst_scene->m_lights, scene->m_lights.data(),
        sizeof(Light) * lightNum, hipMemcpyHostToDevice);

    // Move Primitive Data
    int primitiveNum = scene->m_primitives.size();
    hipMalloc(&hst_scene->m_primitives, sizeof(Primitive) * primitiveNum);
    hipMemcpy(hst_scene->m_primitives, scene->m_primitives.data(),
        sizeof(Primitive) * primitiveNum, hipMemcpyHostToDevice);

    // Move cudaScene Data
    hipMalloc(&dev_scene, sizeof(CUDAScene));
    hipMemcpy(dev_scene, hst_scene, sizeof(CUDAScene), hipMemcpyHostToDevice);

    // Move Camera Data
    hst_camera = &(renderer->m_camera);
    Film& film = hst_camera->m_film;
    hipMalloc(&film.m_bitmap, film.m_resolution.x * film.m_resolution.y * 3 * sizeof(Float));
    hipMemset(film.m_bitmap, 0, film.m_resolution.x * film.m_resolution.y * 3 * sizeof(Float));
    hipMalloc(&film.m_sampleNum, film.m_resolution.x * film.m_resolution.y * sizeof(unsigned int));
    hipMemset(film.m_sampleNum, 0, film.m_resolution.x * film.m_resolution.y * sizeof(unsigned int));
    hipMalloc(&dev_camera, sizeof(Camera));
    hipMemcpy(dev_camera, hst_camera, sizeof(Camera), hipMemcpyHostToDevice);

    // Move Integrator
    hst_integrator = &(renderer->m_integrator);
    hipMalloc(&dev_integrator, sizeof(Integrator));
    hipMemcpy(dev_integrator, hst_integrator, sizeof(Integrator), hipMemcpyHostToDevice);

    hst_renderer = new CUDARenderer(dev_integrator, dev_camera, dev_scene);
    hipMalloc(&dev_renderer, sizeof(CUDARenderer));
    hipMemcpy(dev_renderer, hst_renderer, sizeof(CUDARenderer), hipMemcpyHostToDevice);
}

typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

/*struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float* tnear, float* tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}*/

__device__
Spectrum NextEventEstimate(const CUDAScene& scene, const Interaction& inter, unsigned int& seed, Point3f& pLight) {
    const Primitive& primitive = scene.m_primitives[inter.m_primitiveID];
    const Material& material = scene.m_materials[primitive.m_materialID];

    Spectrum est(0.);
    
    // Sample one of lights
    int lightID = min(scene.m_lightNum - 1, int(NextRandom(seed) * scene.m_lightNum));
    Float lightChoosePdf = Float(1) / scene.m_lightNum;
    const Light& light = scene.m_lights[lightID];

    // Light Sample Li
    const Triangle& triangle = scene.m_triangles[light.m_shapeID];
    Float lightSamplePdf;
    Interaction lightSample = triangle.Sample(&lightSamplePdf, seed);
    pLight = lightSample.m_p;
    lightSamplePdf *= (lightSample.m_p - inter.m_p).SqrLength() / 
        AbsDot(-Normalize(lightSample.m_p - inter.m_p), lightSample.m_shadingN);

    // Visibility test
    Point3f origin = inter.m_p + Normalize(lightSample.m_p - inter.m_p) * Epsilon;
    Point3f target = lightSample.m_p + Normalize(origin - lightSample.m_p) * Epsilon;
    Vector3f d = target - origin;
    Ray testRay(origin, Normalize(d), d.Length() - Epsilon);
    bool hit = scene.Intersect(testRay);


    if (!hit) {
        Vector3f d = Normalize(lightSample.m_p - inter.m_p);
        // Get Le
        Spectrum Le(0.);
        if (Dot(-d, lightSample.m_shadingN) > 0) {
            Le = light.m_L;
        }
        Normal3f n = Faceforward(inter.m_shadingN, d);

        // BSDF
        Spectrum cosineBSDF = material.m_Kd * InvPi * AbsDot(d, n);

        // Contribution
        est = Le * cosineBSDF / lightSamplePdf;
    }
    return est / lightChoosePdf;
}

__device__
Spectrum SampleMaterial(const CUDAScene& scene, Interaction& inter, unsigned int& seed) {
    const Primitive& primitive = scene.m_primitives[inter.m_primitiveID];
    const Material& material = scene.m_materials[primitive.m_materialID];
    
    Spectrum cosBsdf(1.);

    Vector3f wi = CosineSampleHemisphere(seed);
    cosBsdf = material.m_Kd * InvPi * wi.z;
    Float bsdfPdf = CosineSampleHemispherePdf(wi.z);

    Normal3f n = Faceforward(inter.m_shadingN, inter.m_wo);
    Vector3f s, t;
    CoordinateSystem(n, &s, &t);
    inter.m_wi = LocalToWorld(wi, n, s, t);

    return cosBsdf / bsdfPdf;
}

__global__ void
d_render(uint* d_output, uint imageW, uint imageH, int frame, CUDARenderer* renderer)
{
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    Integrator* integrator = renderer->m_integrator;
    Camera* camera = renderer->m_camera;
    CUDAScene* scene = renderer->m_scene;

    uint index = y * imageW + x;
    if ((x >= imageW) || (y >= imageH)) return;

    uint seed = InitRandom(index, frame);
    Spectrum L(0);
    Spectrum throughput(1);
    Ray ray = camera->GenerateRay(Point2f(x + NextRandom(seed), y + NextRandom(seed)));
    for (int i = 0; i < integrator->m_maxDepth; i++) {

        // find intersection with scene
        Interaction interaction;
        bool hit = scene->IntersectP(ray, &interaction);

        if (!hit) {
            break;
        }        

        const Primitive& primitive = scene->m_primitives[interaction.m_primitiveID];
        if (i == 0 && primitive.m_lightID != -1) {
            int lightID = primitive.m_lightID;
            const Light& light = scene->m_lights[lightID];
            if (Dot(interaction.m_shadingN, interaction.m_wo) > 0) {
                L += throughput * light.m_L;
            }
        }

        // render normal
        //L = Spectrum(interaction.m_geometryN);
        //break;

        // get material's bsdf
        const Material& material = scene->m_materials[primitive.m_materialID];        

        // direct light
        Point3f pLight;
        L += throughput * NextEventEstimate(*scene, interaction, seed, pLight);


        // calculate BSDF
        throughput *= SampleMaterial(*scene, interaction, seed);

        // indirect light                    
        if (throughput.Max() < 1 && i > 3) {
            Float q = max((Float).05, 1 - throughput.Max());
            if (NextRandom(seed) < q) break;
            throughput /= 1 - q;
        }

        ray.o = interaction.m_p + interaction.m_wi * Epsilon;
        ray.d = interaction.m_wi;
        ray.tMax = Infinity;
    }
    camera->m_film.AddSample(x, y, L);
    L = camera->m_film.GetPixelSpectrum(index);

    // write output color
    SpectrumToUnsignedChar(L, (unsigned char*)&d_output[(imageH - y - 1) * imageW + x], 4);

}

extern "C"
void freeCudaBuffers()
{
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint * d_output, uint imageW, uint imageH)
{
    d_render << <gridSize, blockSize >> > (d_output, imageW, imageH, frame, dev_renderer);
    frame++;
}

extern "C"
void copyInvViewMatrix(float* invViewMatrix, size_t sizeofMatrix)
{
    //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}


#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
